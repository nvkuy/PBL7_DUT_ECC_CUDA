#include "hip/hip_runtime.h"
﻿#ifdef __HIPCC__
#define CUDA_KERNEL(grid, block, sh_mem, stream) <<< grid, block, sh_mem, stream >>>
#define CUDA_SYNCTHREADS() __syncthreads()
#else
#define CUDA_KERNEL(grid, block, sh_mem, stream)
#define CUDA_SYNCTHREADS()
#define min(a, b) a < b ? a : b
#define max(a, b) a > b ? a : b
#endif

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime_api.h>

#include <iostream>
#include <time.h>
#include <vector>
#include <cassert>
#include <algorithm>
#include <random>
#include <chrono>
#include <queue>
#include <mutex>
#include <condition_variable>

#define CUDA_CHECK(val) check((val), #val, __FILE__, __LINE__)
inline void check(hipError_t err, const char* const func, const char* const file, const int line)
{
	if (err != hipSuccess)
	{
		std::cerr << "CUDA Runtime Error at: " << file << ":" << line << std::endl;
		std::cerr << hipGetErrorString(err) << " " << func << std::endl;
		std::exit(EXIT_FAILURE);
	}
}

#define CUDA_CHECK_LAST() check_last(__FILE__, __LINE__)
inline void check_last(const char* const file, const int line)
{
	hipError_t const err{ hipPeekAtLastError() };
	if (err != hipSuccess)
	{
		std::cerr << "CUDA Runtime Error at: " << file << ":" << line << std::endl;
		std::cerr << hipGetErrorString(err) << std::endl;
		std::exit(EXIT_FAILURE);
	}
}

const unsigned MOD = 65537;
const unsigned SPECIAL = MOD - 1;
const unsigned ROOT = 3;
const unsigned ROOT_INV = 21846;
const unsigned MAX_LOG = 16;

const unsigned LOG_DATA = 10;
const unsigned LOG_SYMBOL = LOG_DATA - 1;
const unsigned LOG_SEG = LOG_SYMBOL - 1;
const unsigned SYMBOL_PER_PACKET = 1 << LOG_SYMBOL;
const unsigned NUM_OF_PACKET = 1 << (MAX_LOG - LOG_SYMBOL);
const unsigned NUM_OF_NEED_PACKET = NUM_OF_PACKET >> 1;
const unsigned SEG_PER_PACKET = 1 << LOG_SEG;
const unsigned SEG_DIFF = 1 << (MAX_LOG - 1);
const unsigned NUM_OF_NEED_SYMBOL = 1 << (MAX_LOG - 1);

const unsigned LEN_ROOT_LAYER_POW = (1 << MAX_LOG) - 1;
const unsigned LEN_ROOT_LAYER_POW_2 = LEN_ROOT_LAYER_POW << 1;
const unsigned LEN_N_POS = ((1 << (MAX_LOG + 1)) - 1);
const unsigned LEN_PACKET_PRODUCT = NUM_OF_PACKET * (SYMBOL_PER_PACKET << 1);
const unsigned LEN_ONE_PACKET_PRODUCT = 1 << (LOG_SYMBOL + 1);

const unsigned LEN_SMALL = NUM_OF_NEED_SYMBOL;
const unsigned LEN_LARGE = LEN_SMALL << 1;

const unsigned SIZE_SMALL = LEN_SMALL * sizeof(unsigned);
const unsigned SIZE_LARGE = LEN_LARGE * sizeof(unsigned);
const unsigned SIZE_ONE_PACKET_PRODUCT = LEN_ONE_PACKET_PRODUCT * sizeof(unsigned);

const unsigned LOG_LEN_ENCODE_P = MAX_LOG - 1;
const unsigned LOG_LEN_ENCODE_Y = MAX_LOG;
const unsigned LOG_LEN_DECODE_X = MAX_LOG - 1;
const unsigned LOG_LEN_DECODE_Y = MAX_LOG - 1;
const unsigned LOG_LEN_DECODE_P = MAX_LOG - 1;

const unsigned LEN_ENCODE_P = 1 << LOG_LEN_ENCODE_P;
const unsigned LEN_ENCODE_Y = 1 << LOG_LEN_ENCODE_Y;
const unsigned LEN_DECODE_X = 1 << LOG_LEN_DECODE_X;
const unsigned LEN_DECODE_Y = 1 << LOG_LEN_DECODE_Y;
const unsigned LEN_DECODE_P = 1 << LOG_LEN_DECODE_P;

const unsigned SIZE_ENCODE_P = LEN_ENCODE_P * sizeof(unsigned);
const unsigned SIZE_ENCODE_Y = LEN_ENCODE_Y * sizeof(unsigned);
const unsigned SIZE_DECODE_X = LEN_DECODE_X * sizeof(unsigned);
const unsigned SIZE_DECODE_Y = LEN_DECODE_Y * sizeof(unsigned);
const unsigned SIZE_DECODE_P = LEN_DECODE_P * sizeof(unsigned);

//const unsigned SM_CNT = 28;
//const unsigned MAX_WARP = 48 * 4 * SM_CNT;

const unsigned MAX_ACTIVE_ENCODE = 256;
const unsigned MAX_ACTIVE_DECODE = 256;
const unsigned MAX_ENCODE_LAUNCH_CNT = 16;
const unsigned MAX_DECODE_LAUNCH_CNT = 256;

const unsigned LOG_THREAD_PER_OP = 9;
const unsigned THREAD_PER_OP = 1 << LOG_THREAD_PER_OP;
//const unsigned N_BL = 1, N_TH = THREAD_PER_OP;

const unsigned LOG_LEN_WARP = 5;
const unsigned LEN_WARP = 1 << LOG_LEN_WARP;
const unsigned ALGO_N_2_CUTOFF = 64;

const size_t SIZE_ENCODE_P_SLOT = sizeof(unsigned) * LEN_ENCODE_P * MAX_ACTIVE_ENCODE;
const size_t SIZE_ENCODE_Y_SLOT = sizeof(unsigned) * LEN_ENCODE_Y * MAX_ACTIVE_ENCODE;

const size_t SIZE_DECODE_X_SLOT = sizeof(unsigned) * LEN_DECODE_X * MAX_ACTIVE_DECODE;
const size_t SIZE_DECODE_Y_SLOT = sizeof(unsigned) * LEN_DECODE_Y * MAX_ACTIVE_DECODE;
//const size_t SIZE_DECODE_P_SLOT = sizeof(unsigned) * LEN_DECODE_P * MAX_ACTIVE_DECODE;
const size_t SIZE_DECODE_T1_SLOT = sizeof(unsigned) * LEN_LARGE * MAX_ACTIVE_DECODE;
const size_t SIZE_DECODE_T2_SLOT = sizeof(unsigned) * LEN_LARGE * MAX_ACTIVE_DECODE;
const size_t SIZE_DECODE_AX_SLOT = sizeof(unsigned) * LEN_LARGE * MAX_ACTIVE_DECODE;
const size_t SIZE_DECODE_DAX_SLOT = sizeof(unsigned) * LEN_SMALL * MAX_ACTIVE_DECODE;
const size_t SIZE_DECODE_VDAX_SLOT = sizeof(unsigned) * LEN_LARGE * MAX_ACTIVE_DECODE;
const size_t SIZE_DECODE_N1_SLOT = sizeof(unsigned) * LEN_SMALL * MAX_ACTIVE_DECODE;
const size_t SIZE_DECODE_N2_SLOT = sizeof(unsigned) * LEN_LARGE * MAX_ACTIVE_DECODE;
const size_t SIZE_DECODE_N3_SLOT = sizeof(unsigned) * LEN_SMALL * MAX_ACTIVE_DECODE;

unsigned** h_encode_p_slot;
unsigned** h_encode_y_slot;
unsigned** h_decode_x_slot;
unsigned** h_decode_y_slot;
unsigned** h_decode_p_slot;

unsigned* d_encode_p_slot;
unsigned* d_encode_y_slot;
unsigned* d_decode_x_slot;
unsigned* d_decode_y_slot;
//unsigned* d_decode_p_slot;
unsigned* d_decode_t1_slot;
unsigned* d_decode_t2_slot;
unsigned* d_decode_ax_slot;
unsigned* d_decode_dax_slot;
unsigned* d_decode_vdax_slot;
unsigned* d_decode_n1_slot;
unsigned* d_decode_n2_slot;
unsigned* d_decode_n3_slot;

unsigned* d_N_pos;
unsigned* d_root_pow;
unsigned* d_root_inv_pow;
unsigned* d_inv;
unsigned* d_root_layer_pow;
unsigned* d_packet_product;

struct CB_DATA {
	unsigned slot_id;
	unsigned* dst;
	unsigned* src;
	size_t size_res;
	std::queue<unsigned>& slot; 
	std::mutex& mt; 
	std::condition_variable& cv;
};

std::queue<unsigned> encode_slot, decode_slot;
std::mutex mt_encode_slot, mt_decode_slot;
std::condition_variable cv_encode_slot, cv_decode_slot;

inline unsigned pop_slot(std::queue<unsigned> &slot, std::mutex &mt, std::condition_variable &cv) {
	std::unique_lock<std::mutex> lock(mt);
	cv.wait(lock, [&] { return !slot.empty(); });
	unsigned id = slot.front();
	slot.pop();
	return id;
}

inline void push_slot(unsigned id, std::queue<unsigned>& slot, std::mutex& mt, std::condition_variable& cv) {
	{
		std::lock_guard<std::mutex> lock(mt);
		slot.push(id);
	}
	cv.notify_one();
}

void init_batch_slot() {

	for (unsigned i = 0; i < MAX_ACTIVE_ENCODE; i++)
		push_slot(i, encode_slot, mt_encode_slot, cv_encode_slot);

	for (unsigned i = 0; i < MAX_ACTIVE_DECODE; i++)
		push_slot(i, decode_slot, mt_decode_slot, cv_decode_slot);

}

void CUDART_CB h_end_batch_slot(void* data) {

	CB_DATA* dat = static_cast<CB_DATA*>(data);

	memcpy(dat->dst, dat->src, dat->size_res);
	push_slot(dat->slot_id, dat->slot, dat->mt, dat->cv);

	delete dat;
}

__host__ __device__ __forceinline__ unsigned mul_mod(unsigned a, unsigned b)
{
	if (a == SPECIAL && b == SPECIAL)
		return 1; // overflow
	return (a * b) % MOD;
}

__device__ __forceinline__ unsigned div_mod(unsigned a, unsigned b,
	unsigned* d_inv)
{
	return mul_mod(a, d_inv[b]);
}

__host__ __device__ __forceinline__ unsigned add_mod(unsigned a, unsigned b)
{
	return (a + b) % MOD;
}

__host__ __device__ __forceinline__ unsigned sub_mod(unsigned a, unsigned b)
{
	return (a - b + MOD) % MOD;
}

__host__ __device__ __forceinline__ unsigned pow_mod(unsigned a, unsigned b)
{
	unsigned res = 1;
	while (b > 0)
	{
		if (b & 1)
			res = mul_mod(res, a);
		a = mul_mod(a, a);
		b >>= 1;
	}
	return res;
}

__global__ void fnt(unsigned* a, unsigned* b, unsigned log_na, unsigned log_nb, unsigned opt,
	unsigned* d_N_pos, unsigned* d_root_layer_pow, unsigned* d_inv, unsigned wpt)
{

	/*
	opt 2 bit: x1 x2
	 - x1: w_n or 1/w_n
	 - x2: need result * 1/n
	*/

	// size_b >= size_a;
	// need memset *b before use unless size_a == size_b

	// have size_b/2 tasks

	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned na = 1 << log_na, nb = 1 << log_nb, wp = (opt & 2) >> 1, st = nb - 1;
	unsigned id_l = id * wpt, id_r = min(id_l + wpt, nb >> 1);
	unsigned os1 = nb >> 1, os2 = LEN_ROOT_LAYER_POW * wp;
	for (unsigned k = id_l; k < id_r; k++) {
		if (k < na) b[d_N_pos[st + k]] = a[k];
		if (log_na == log_nb) b[d_N_pos[st + k + os1]] = a[k + os1];
	}

	CUDA_SYNCTHREADS();

	for (unsigned i = 0; i < log_nb; i++) {

		unsigned haft_len = 1 << i;
		for (unsigned k = id_l; k < id_r; k++) {
			unsigned bl_st = ((k >> i) << (i + 1)), th_id = (k & (haft_len - 1));
			unsigned pos = bl_st + th_id;
			unsigned u = b[pos];
			unsigned v = mul_mod(b[pos + haft_len], d_root_layer_pow[os2 + haft_len - 1 + th_id]);
			b[pos] = add_mod(u, v);
			b[pos + haft_len] = sub_mod(u, v);
		}

		CUDA_SYNCTHREADS();

	}

	if (opt & 1) {
		for (unsigned k = id_l; k < id_r; k++) {
			b[k] = div_mod(b[k], nb, d_inv);
			b[k + os1] = div_mod(b[k + os1], nb, d_inv);
		}

	}
}

__global__ void g_vector_mul_i(unsigned* a, unsigned* b, unsigned* c, unsigned n, unsigned wpt)
{

	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned id_l = id * wpt, id_r = min(id_l + wpt, n);
	for (unsigned k = id_l; k < id_r; k++)
		c[k] = mul_mod(a[k], b[k]);


}

__global__ void g_fill(unsigned* a, unsigned val, unsigned n, unsigned wpt) {

	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned id_l = id * wpt, id_r = min(id_l + wpt, n);
	for (unsigned k = id_l; k < id_r; k++)
		a[k] = val;

}

__global__ void g_cpy(unsigned* a, unsigned* b, unsigned n, unsigned wpt) {

	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
		unsigned id_l = id * wpt, id_r = min(id_l + wpt, n);
		for (unsigned k = id_l; k < id_r; k++)
			b[k] = a[k];

}

inline void h_poly_mul(unsigned* a, unsigned* b, unsigned* t1, unsigned* t2, unsigned* c, unsigned log_n,
	unsigned* d_N_pos, unsigned* d_root_layer_pow, unsigned* d_inv, hipStream_t stream) {


	unsigned na = 1 << log_n, nc = 1 << (log_n + 1), size_nc = nc * sizeof(unsigned);

	unsigned wpt1 = max(nc >> LOG_THREAD_PER_OP, 1), wpt2 = max(na >> LOG_THREAD_PER_OP, 1);
	g_fill CUDA_KERNEL(1, THREAD_PER_OP, NULL, stream) (t1, 0, nc, wpt1);
	g_fill CUDA_KERNEL(1, THREAD_PER_OP, NULL, stream) (t2, 0, nc, wpt1);

	fnt CUDA_KERNEL(1, THREAD_PER_OP, NULL, stream) (a, t1, log_n, log_n + 1, 0, d_N_pos, d_root_layer_pow, d_inv, wpt2);
	fnt CUDA_KERNEL(1, THREAD_PER_OP, NULL, stream) (b, t2, log_n, log_n + 1, 0, d_N_pos, d_root_layer_pow, d_inv, wpt2);

	g_vector_mul_i CUDA_KERNEL(1, THREAD_PER_OP, NULL, stream)(t1, t2, t1, nc, wpt1);

	fnt CUDA_KERNEL(1, THREAD_PER_OP, NULL, stream) (t1, c, log_n + 1, log_n + 1, 3, d_N_pos, d_root_layer_pow, d_inv, wpt2);

}

__global__ void g_poly_deriv(unsigned* ax, unsigned* dax, unsigned wpt)
{

	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned id_l = id * wpt, id_r = id_l + wpt;
	for (unsigned k = id_l; k < id_r; k++)
		dax[k] = mul_mod(ax[k + 1], k + 1);
}

inline void h_build_product(unsigned* p, unsigned* t1, unsigned* t2, unsigned log_n1, unsigned log_n2, hipStream_t stream)
{

	// p, t1, t2 in device

	for (unsigned i = log_n1; i < log_n2; i++)
	{
		unsigned m = 1 << (log_n2 - i - 1), len = 1 << i;
		for (unsigned j = 0; j < m; j++) {
			unsigned st = j << (i + 1);
			h_poly_mul(p + st, p + st + len, t1 + st, t2 + st, p + st, i, d_N_pos, d_root_layer_pow, d_inv, stream);
		}
	}
}

inline void h_build_ax(unsigned* x, unsigned* p, unsigned* t1, unsigned* t2, hipStream_t stream)
{

	// p, t1, t2 in device
	// x in host

	for (unsigned i = 0; i < NUM_OF_NEED_PACKET; i++)
	{
		unsigned st_p1 = i << (LOG_SYMBOL + 1), st_p2 = x[i << LOG_SYMBOL] << 2;
		//CUDA_CHECK(hipMemcpyAsync(p + st_p1, d_packet_product + st_p2, SIZE_ONE_PACKET_PRODUCT, hipMemcpyDeviceToDevice, stream));
		g_cpy CUDA_KERNEL(1, THREAD_PER_OP, NULL, stream) (d_packet_product + st_p2, p + st_p1, LEN_ONE_PACKET_PRODUCT, LEN_ONE_PACKET_PRODUCT >> LOG_THREAD_PER_OP);
		CUDA_CHECK_LAST();
	}
	h_build_product(p, t1, t2, LOG_SYMBOL + 1, MAX_LOG, stream);
}

__global__ void g_build_n1(unsigned* n1, unsigned* vdax, unsigned* x, unsigned* y,
	unsigned* d_inv, unsigned wpt)
{

	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned id_l = id * wpt, id_r = id_l + wpt;
	for (unsigned k = id_l; k < id_r; k++)
		n1[k] = div_mod(y[k], vdax[x[k]], d_inv);

}

__global__ void g_build_n2(unsigned* n2, unsigned* n1, unsigned* x, unsigned wpt) {

	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned id_l = id * wpt, id_r = id_l + wpt;
	for (unsigned k = id_l; k < id_r; k++)
		n2[x[k]] = n1[k];

}

__global__ void g_build_n3(unsigned* n3, unsigned* p_n3, unsigned wpt) {

	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned id_l = id * wpt, id_r = id_l + wpt;
	for (unsigned k = id_l; k < id_r; k++)
		n3[k] = sub_mod(0, p_n3[k + 1]);

}

inline void h_build_px(unsigned* p, unsigned* ax, unsigned* n3, unsigned* t1, unsigned* t2,
	unsigned* d_N_pos, unsigned* d_root_layer_pow, unsigned* d_inv, hipStream_t stream) {

	h_poly_mul(ax, n3, t1, t2, p, MAX_LOG - 1, d_N_pos, d_root_layer_pow, d_inv, stream);
	CUDA_CHECK_LAST();

}

void h_encode(unsigned* p, unsigned* y)
{

	unsigned slot_id = pop_slot(encode_slot, mt_encode_slot, cv_encode_slot);

	unsigned* sl_p = h_encode_p_slot[slot_id];
	unsigned* sl_y = h_encode_y_slot[slot_id];

	memcpy(sl_p, p, SIZE_ENCODE_P);

	unsigned* d_p = d_encode_p_slot + 1LL * slot_id * LEN_ENCODE_P;
	unsigned* d_y = d_encode_y_slot + 1LL * slot_id * LEN_ENCODE_Y;

	unsigned wpt1 = LEN_LARGE >> LOG_THREAD_PER_OP, wpt2 = LEN_SMALL >> LOG_THREAD_PER_OP;

	CB_DATA* data = new CB_DATA{ slot_id, y, sl_y, SIZE_ENCODE_Y, encode_slot, mt_encode_slot, cv_encode_slot };

	hipStream_t stream;
	CUDA_CHECK(hipStreamCreate(&stream));

	CUDA_CHECK(hipMemcpyAsync(d_p, sl_p, SIZE_ENCODE_P, hipMemcpyHostToDevice, stream));
	
	g_fill CUDA_KERNEL(1, THREAD_PER_OP, NULL, stream) (d_y, 0, LEN_LARGE, wpt1);

	fnt CUDA_KERNEL(1, THREAD_PER_OP, NULL, stream) (d_p, d_y, MAX_LOG - 1, MAX_LOG, 0, d_N_pos, d_root_layer_pow, d_inv, wpt2);
	CUDA_CHECK_LAST();

	CUDA_CHECK(hipMemcpyAsync(sl_y, d_y, SIZE_ENCODE_Y, hipMemcpyDeviceToHost, stream));
	
	CUDA_CHECK(hipLaunchHostFunc(stream, h_end_batch_slot, data));

	CUDA_CHECK(hipStreamDestroy(stream));

}

void h_decode(unsigned* x, unsigned* y, unsigned* p)
{

	unsigned slot_id = pop_slot(decode_slot, mt_decode_slot, cv_decode_slot);

	unsigned* sl_x = h_decode_x_slot[slot_id];
	unsigned* sl_y = h_decode_y_slot[slot_id];
	unsigned* sl_p = h_decode_p_slot[slot_id];

	memcpy(sl_x, x, SIZE_DECODE_X);
	memcpy(sl_y, y, SIZE_DECODE_Y);

	unsigned* d_x = d_decode_x_slot + 1LL * slot_id * LEN_DECODE_X;
	unsigned* d_y = d_decode_y_slot + 1LL * slot_id * LEN_DECODE_Y;
	unsigned* d_t1 = d_decode_t1_slot + 1LL * slot_id * LEN_LARGE;
	unsigned* d_t2 = d_decode_t2_slot + 1LL * slot_id * LEN_LARGE;
	unsigned* d_ax = d_decode_ax_slot + 1LL * slot_id * LEN_LARGE;
	unsigned* d_dax = d_decode_dax_slot + 1LL * slot_id * LEN_SMALL;
	unsigned* d_vdax = d_decode_vdax_slot + 1LL * slot_id * LEN_LARGE;
	unsigned* d_n1 = d_decode_n1_slot + 1LL * slot_id * LEN_SMALL;
	unsigned* d_n2 = d_decode_n2_slot + 1LL * slot_id * LEN_LARGE;
	unsigned* d_n3 = d_decode_n3_slot + 1LL * slot_id * LEN_SMALL;

	unsigned wpt1 = LEN_LARGE >> LOG_THREAD_PER_OP, wpt2 = LEN_SMALL >> LOG_THREAD_PER_OP;

	CB_DATA* data = new CB_DATA{ slot_id, p, sl_p, SIZE_DECODE_P, decode_slot, mt_decode_slot, cv_decode_slot };

	hipStream_t stream;
	CUDA_CHECK(hipStreamCreate(&stream));

	CUDA_CHECK(hipMemcpyAsync(d_x, sl_x, SIZE_DECODE_X, hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(d_y, sl_y, SIZE_DECODE_Y, hipMemcpyHostToDevice, stream));

	g_fill CUDA_KERNEL(1, THREAD_PER_OP, NULL, stream)(d_vdax, 0, LEN_LARGE, wpt1);
	g_fill CUDA_KERNEL(1, THREAD_PER_OP, NULL, stream)(d_n2, 0, LEN_LARGE, wpt1);

	h_build_ax(x, d_ax, d_t1, d_t2, stream);

	g_poly_deriv CUDA_KERNEL(1, THREAD_PER_OP, NULL, stream)(d_ax, d_dax, wpt2);
	CUDA_CHECK_LAST();
	
	fnt CUDA_KERNEL(1, THREAD_PER_OP, NULL, stream)(d_dax, d_vdax, MAX_LOG - 1, MAX_LOG, 0, d_N_pos, d_root_layer_pow, d_inv, wpt2);
	CUDA_CHECK_LAST();
	
	g_build_n1 CUDA_KERNEL(1, THREAD_PER_OP, NULL, stream)(d_n1, d_vdax, d_x, d_y, d_inv, wpt2);
	CUDA_CHECK_LAST();
	
	g_build_n2 CUDA_KERNEL(1, THREAD_PER_OP, NULL, stream)(d_n2, d_n1, d_x, wpt2);
	CUDA_CHECK_LAST();
	
	fnt CUDA_KERNEL(1, THREAD_PER_OP, NULL, stream)(d_n2, d_t2, MAX_LOG, MAX_LOG, 2, d_N_pos, d_root_layer_pow, d_inv, wpt2);
	CUDA_CHECK_LAST();
	g_build_n3 CUDA_KERNEL(1, THREAD_PER_OP, NULL, stream)(d_n3, d_t2, wpt2);
	CUDA_CHECK_LAST();
	
	h_build_px(d_n2, d_ax, d_n3, d_t1, d_t2, d_N_pos, d_root_layer_pow, d_inv, stream);
	CUDA_CHECK(hipMemcpyAsync(sl_p, d_n2, SIZE_SMALL, hipMemcpyDeviceToHost, stream));

	CUDA_CHECK(hipLaunchHostFunc(stream, h_end_batch_slot, data));

	CUDA_CHECK(hipStreamDestroy(stream));

}

void init()
{
	// offline process

	CUDA_CHECK(hipDeviceSynchronize());
	CUDA_CHECK(hipDeviceReset());
	//CUDA_CHECK(hipDeviceSetLimit(cudaLimitDevRuntimePendingLaunchCount, 
	//	max(MAX_ACTIVE_DECODE * MAX_DECODE_LAUNCH_CNT, MAX_ACTIVE_ENCODE * MAX_ENCODE_LAUNCH_CNT)));

	//hipDeviceProp_t prop;
    //int device;
    //hipGetDevice(&device);
    //hipGetDeviceProperties(&prop, device);
    //std::cout << "Device name: " << prop.name << std::endl;
	//std::cout << "Allow concurrent Kernels: " << prop.concurrentKernels << std::endl;
    //std::cout << "Max threads per block: " << prop.maxThreadsPerBlock << std::endl;
    //std::cout << "Max threads per SM: " << prop.maxThreadsPerMultiProcessor << std::endl;
    //std::cout << "Number of SMs: " << prop.multiProcessorCount << std::endl;
    //std::cout << "Max concurrent threads on device: " << prop.maxThreadsPerMultiProcessor * prop.multiProcessorCount << std::endl;

	h_encode_p_slot = (unsigned**)malloc(MAX_ACTIVE_ENCODE * sizeof(unsigned*));
	h_encode_y_slot = (unsigned**)malloc(MAX_ACTIVE_ENCODE * sizeof(unsigned*));
	h_decode_x_slot = (unsigned**)malloc(MAX_ACTIVE_DECODE * sizeof(unsigned*));
	h_decode_y_slot = (unsigned**)malloc(MAX_ACTIVE_DECODE * sizeof(unsigned*));
	h_decode_p_slot = (unsigned**)malloc(MAX_ACTIVE_DECODE * sizeof(unsigned*));

	for (unsigned i = 0; i < MAX_ACTIVE_ENCODE; i++) {
		CUDA_CHECK(hipHostMalloc(&(h_encode_p_slot[i]), SIZE_ENCODE_P));
		CUDA_CHECK(hipHostMalloc(&(h_encode_y_slot[i]), SIZE_ENCODE_Y));
	}

	for (unsigned i = 0; i < MAX_ACTIVE_DECODE; i++) {
		CUDA_CHECK(hipHostMalloc(&(h_decode_x_slot[i]), SIZE_DECODE_X));
		CUDA_CHECK(hipHostMalloc(&(h_decode_y_slot[i]), SIZE_DECODE_Y));
		CUDA_CHECK(hipHostMalloc(&(h_decode_p_slot[i]), SIZE_DECODE_P));
	}

	CUDA_CHECK(hipMalloc(&d_encode_p_slot, SIZE_ENCODE_P_SLOT));
	CUDA_CHECK(hipMalloc(&d_encode_y_slot, SIZE_ENCODE_Y_SLOT));

	CUDA_CHECK(hipMalloc(&d_decode_x_slot, SIZE_DECODE_X_SLOT));
	CUDA_CHECK(hipMalloc(&d_decode_y_slot, SIZE_DECODE_Y_SLOT));
	//CUDA_CHECK(hipMalloc(&d_decode_p_slot, SIZE_DECODE_P_SLOT));
	CUDA_CHECK(hipMalloc(&d_decode_t1_slot, SIZE_DECODE_T1_SLOT));
	CUDA_CHECK(hipMalloc(&d_decode_t2_slot, SIZE_DECODE_T2_SLOT));
	CUDA_CHECK(hipMalloc(&d_decode_ax_slot, SIZE_DECODE_AX_SLOT));
	CUDA_CHECK(hipMalloc(&d_decode_dax_slot, SIZE_DECODE_DAX_SLOT));
	CUDA_CHECK(hipMalloc(&d_decode_vdax_slot, SIZE_DECODE_VDAX_SLOT));
	CUDA_CHECK(hipMalloc(&d_decode_n1_slot, SIZE_DECODE_N1_SLOT));
	CUDA_CHECK(hipMalloc(&d_decode_n2_slot, SIZE_DECODE_N2_SLOT));
	CUDA_CHECK(hipMalloc(&d_decode_n3_slot, SIZE_DECODE_N3_SLOT));

	init_batch_slot();

	unsigned size_N_pos = LEN_N_POS * sizeof(unsigned);
	unsigned* N_pos = (unsigned*)malloc(size_N_pos);
	CUDA_CHECK(hipMalloc(&d_N_pos, size_N_pos));

	for (unsigned i = 1; i <= MAX_LOG; i++)
	{
		unsigned n = 1 << i, st = n - 1;
		for (unsigned j = 0; j < n; j++)
			N_pos[st + j] = j;
	}

	for (unsigned i = 1; i <= MAX_LOG; i++)
	{
		unsigned n = 1 << i, st = n - 1;
		for (unsigned j = 0; j < n; j++)
		{
			unsigned rev_num = 0;
			for (unsigned k = 0; k < i; k++)
			{
				if (j & (1 << k))
					rev_num |= (1 << (i - 1 - k));
			}
			if (j < rev_num)
				std::swap(N_pos[st + j], N_pos[st + rev_num]);
		}
	}

	CUDA_CHECK(hipMemcpy(d_N_pos, N_pos, size_N_pos, hipMemcpyHostToDevice));
	free(N_pos);

	unsigned size_root = MOD * sizeof(unsigned);
	unsigned* root_pow = (unsigned*)malloc(size_root);
	unsigned* root_inv_pow = (unsigned*)malloc(size_root);
	unsigned* inv = (unsigned*)malloc(size_root);
	CUDA_CHECK(hipMalloc(&d_root_pow, size_root));
	CUDA_CHECK(hipMalloc(&d_root_inv_pow, size_root));
	CUDA_CHECK(hipMalloc(&d_inv, size_root));

	root_pow[0] = 1, root_inv_pow[0] = 1, inv[0] = 0;
	for (unsigned i = 1; i < MOD; i++)
	{
		root_pow[i] = mul_mod(root_pow[i - 1], ROOT);
		root_inv_pow[i] = mul_mod(root_inv_pow[i - 1], ROOT_INV);
		inv[i] = pow_mod(i, MOD - 2);
	}

	CUDA_CHECK(hipMemcpy(d_root_pow, root_pow, size_root, hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(d_root_inv_pow, root_inv_pow, size_root, hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(d_inv, inv, size_root, hipMemcpyHostToDevice));

	unsigned size_root_layer_pow = LEN_ROOT_LAYER_POW_2 * sizeof(unsigned);
	unsigned* root_layer_pow = (unsigned*)malloc(size_root_layer_pow);
	CUDA_CHECK(hipMalloc(&d_root_layer_pow, size_root_layer_pow));

	for (unsigned i = 0; i < 2; i++)
	{
		unsigned st_i = LEN_ROOT_LAYER_POW * i;
		for (unsigned j = 0; j < MAX_LOG; j++)
		{
			unsigned haft_len = 1 << j;
			unsigned st_j = haft_len - 1;
			unsigned ang = 1 << (MAX_LOG - j - 1);
			unsigned wn = i ? root_inv_pow[ang] : root_pow[ang], w = 1;
			for (unsigned k = 0; k < haft_len; k++)
			{
				root_layer_pow[st_i + st_j + k] = w;
				w = mul_mod(w, wn);
			}
		}
	}

	CUDA_CHECK(hipMemcpy(d_root_layer_pow, root_layer_pow, size_root_layer_pow, hipMemcpyHostToDevice));
	free(root_layer_pow);

	unsigned size_packet_product = LEN_PACKET_PRODUCT * sizeof(unsigned);
	unsigned* packet_product = (unsigned*)malloc(size_packet_product);
	CUDA_CHECK(hipMalloc(&d_packet_product, size_packet_product));

	for (unsigned i = 0; i < NUM_OF_PACKET; i++)
	{
		unsigned st = i << (LOG_SYMBOL + 1);
		for (unsigned j = 0; j < SEG_PER_PACKET; j++)
		{
			unsigned k = (i << LOG_SEG) + j;
			packet_product[st + (j << 1)] = sub_mod(0, root_pow[k]);
			packet_product[st + ((j << 1) | 1)] = 1;
			packet_product[st + ((j + SEG_PER_PACKET) << 1)] = sub_mod(0, root_pow[k + SEG_DIFF]);
			packet_product[st + (((j + SEG_PER_PACKET) << 1) | 1)] = 1;
		}
	}
	CUDA_CHECK(hipMemcpy(d_packet_product, packet_product, size_packet_product, hipMemcpyHostToDevice));
	free(packet_product);
	unsigned* tmp;
	CUDA_CHECK(hipMalloc(&tmp, (LEN_ONE_PACKET_PRODUCT << 1) * sizeof(unsigned)));
	for (unsigned i = 0; i < NUM_OF_PACKET; i++)
	{
		unsigned st = i << (LOG_SYMBOL + 1);
		h_build_product(d_packet_product + st, tmp, tmp + LEN_ONE_PACKET_PRODUCT, 1, LOG_SYMBOL + 1, NULL);
	}
	CUDA_CHECK(hipFree(tmp));
	free(inv);
	free(root_pow);
	free(root_inv_pow);

	CUDA_CHECK(hipDeviceSynchronize());
	std::cout << "Init process completed!" << std::endl;

}

void fin()
{
	// clear cuda memory

	CUDA_CHECK(hipDeviceSynchronize());

	for (unsigned i = 0; i < MAX_ACTIVE_ENCODE; i++) {
		CUDA_CHECK(hipHostFree(h_encode_p_slot[i]));
		CUDA_CHECK(hipHostFree(h_encode_y_slot[i]));
	}

	for (unsigned i = 0; i < MAX_ACTIVE_DECODE; i++) {
		CUDA_CHECK(hipHostFree(h_decode_x_slot[i]));
		CUDA_CHECK(hipHostFree(h_decode_y_slot[i]));
		CUDA_CHECK(hipHostFree(h_decode_p_slot[i]));
	}

	free(h_encode_p_slot);
	free(h_encode_y_slot);
	free(h_decode_x_slot);
	free(h_decode_y_slot);
	free(h_decode_p_slot);

	CUDA_CHECK_LAST();

	CUDA_CHECK(hipDeviceReset());
	CUDA_CHECK(hipDeviceSynchronize());

	CUDA_CHECK_LAST();

}

void test_fnt();

void test_poly_mul();

void test_build_init_product();

void test_encode_decode();

void test_fnt_performance();

void test_encode_decode_performance();

int main()
{

	init();

	test_fnt();
	
	test_poly_mul();
	
	test_build_init_product();
	
	test_encode_decode();
	
	//test_fnt_performance();
	
	test_encode_decode_performance();

	fin();

	return 0;
}

void test_fnt() {

	// test correctness of fnt()

	unsigned N_test = 32;

	for (unsigned tt = 0; tt < N_test; tt++) {
		unsigned log_nc = 15, log_nv = 16, nc = 1 << log_nc, nv = 1 << log_nv;
		unsigned size_nc = nc * sizeof(unsigned), size_nv = nv * sizeof(unsigned);
		std::vector<unsigned> c1(nc), c2(nc);
		unsigned* d_c1, * d_c2, * d_v;
		hipMalloc(&d_c1, size_nc);
		hipMemset(d_c1, 0, size_nc);
		hipMalloc(&d_c2, size_nv);
		hipMemset(d_c2, 0, size_nv);
		hipMalloc(&d_v, size_nv);
		hipMemset(d_v, 0, size_nv);

		for (unsigned i = 0; i < nc; i++)
			c1[i] = rand() % (MOD - 1);
		shuffle(c1.begin(), c1.end(), std::default_random_engine(time(NULL)));
		hipMemcpy(d_c1, c1.data(), size_nc, hipMemcpyHostToDevice);


		unsigned wpt = nv >> (LOG_THREAD_PER_OP + 1);
		fnt CUDA_KERNEL(1, THREAD_PER_OP, NULL, NULL)(d_c1, d_v, log_nc, log_nv, 0, d_N_pos, d_root_layer_pow, d_inv, wpt);
		fnt CUDA_KERNEL(1, THREAD_PER_OP, NULL, NULL)(d_v, d_c2, log_nv, log_nv, 3, d_N_pos, d_root_layer_pow, d_inv, wpt);

		hipMemcpy(c2.data(), d_c2, size_nc, hipMemcpyDeviceToHost);
		for (unsigned i = 0; i < nc; i++)
			assert(c1[i] == c2[i]);

		hipFree(d_c1);
		hipFree(d_c2);
		hipFree(d_v);

		//std::cout << "FNT test " << tt << " passed!" << std::endl;

	}

	std::cout << "FNT test passed!" << std::endl;

	CUDA_CHECK_LAST();

}

void test_build_init_product() {

	// first 10 element..
	std::vector<unsigned> a1 = { 64375, 0, 52012, 0, 2347, 0, 23649, 0, 30899, 0 }, b1(10);
	hipMemcpy(b1.data(), d_packet_product, 10 * sizeof(unsigned), hipMemcpyDeviceToHost);

	for (unsigned i = 0; i < 10; i++)
		assert(a1[i] == b1[i]);

	// first 10 element of next packet..
	std::vector<unsigned> a2 = { 64375, 0, 31561, 0, 12153, 0, 31103, 0, 20714, 0 }, b2(10);
	hipMemcpy(b2.data(), d_packet_product + (1 << (LOG_SYMBOL + 1)), 10 * sizeof(unsigned), hipMemcpyDeviceToHost);

	for (unsigned i = 0; i < 10; i++)
		assert(a2[i] == b2[i]);

	std::cout << "Test packet_product passed!" << std::endl;

	CUDA_CHECK_LAST();

}

void test_poly_mul() {

	// test correctness of poly_mul()

	srand(time(NULL));

	unsigned N_test = 32;

	for (unsigned tt = 0; tt < N_test; tt++) {

		unsigned log_n = 11;
		unsigned n = 1 << log_n, size_n = n * sizeof(unsigned);

		std::vector<unsigned> a(n), b(n), c1(n << 1, 0), c2(n << 1, 0);

		for (unsigned i = 0; i < n; i++) {
			a[i] = rand() % (MOD - 1); // 2 bytes
			b[i] = rand() % (MOD - 1); // 2 bytes
		}

		unsigned* t1, * t2, * d_c;
		hipMalloc(&t1, size_n << 1);
		hipMalloc(&t2, size_n << 1);
		hipMalloc(&d_c, size_n << 1);
		hipMemcpy(d_c, a.data(), size_n, hipMemcpyHostToDevice);
		hipMemcpy(d_c + n, b.data(), size_n, hipMemcpyHostToDevice);
		h_poly_mul(d_c, d_c + n, t1, t2, d_c, log_n, d_N_pos, d_root_layer_pow, d_inv, NULL);

		for (unsigned i = 0; i < n; i++)
			for (unsigned j = 0; j < n; j++)
				c1[i + j] = add_mod(c1[i + j], mul_mod(a[i], b[j]));

		/*unsigned* d_a, * d_b;
		hipMalloc(&d_a, size_n);
		hipMalloc(&d_b, size_n);
		hipMemcpy(d_a, a.data(), size_n, hipMemcpyHostToDevice);
		hipMemcpy(d_b, b.data(), size_n, hipMemcpyHostToDevice);
		poly_mul_wrapper CUDA_KERNEL(1, 1, NULL, NULL)(d_a, d_b, t1, t2, d_c, log_n, d_N_pos, d_root_layer_pow, d_inv);*/

		hipMemcpy(c2.data(), d_c, size_n << 1, hipMemcpyDeviceToHost);

		for (unsigned i = 0; i < (n << 1); i++)
			assert(c1[i] == c2[i]);

		//std::cout << "Poly mul test " << tt << " passed!" << std::endl;

		hipFree(t1);
		hipFree(t2);
		hipFree(d_c);

	}

	std::cout << "Poly mul test passed!" << std::endl;

	CUDA_CHECK_LAST();

}

void test_encode_decode() {

	// test correctness of encode() and decode()

	srand(time(NULL));

	unsigned N_test = 32;

	for (unsigned tt = 0; tt < N_test; tt++) {
		std::vector<unsigned> a(NUM_OF_NEED_SYMBOL), b(NUM_OF_NEED_SYMBOL << 1), c(NUM_OF_NEED_SYMBOL);

		for (unsigned i = 0; i < NUM_OF_NEED_SYMBOL; i++)
			a[i] = rand() % (MOD - 1); // 2 bytes

		h_encode(a.data(), b.data());
		hipDeviceSynchronize();

		std::vector<unsigned> x(NUM_OF_NEED_SYMBOL), y(NUM_OF_NEED_SYMBOL);

		for (unsigned i = 0; i < NUM_OF_NEED_PACKET; i++) {
			unsigned stx = i * SYMBOL_PER_PACKET;
			for (unsigned j = 0; j < SEG_PER_PACKET; j++) {
				x[stx + j] = stx + j;
				x[stx + j + SEG_PER_PACKET] = stx + j + SEG_DIFF;
				y[stx + j] = b[stx + j];
				y[stx + j + SEG_PER_PACKET] = b[stx + j + SEG_DIFF];
			}
		}

		h_decode(x.data(), y.data(), c.data());
		hipDeviceSynchronize();

		for (unsigned i = 0; i < NUM_OF_NEED_SYMBOL; i++)
			assert(a[i] == c[i]);
		//std::cout << "Encode decode test " << tt << " passed!" << std::endl;
	}

	std::cout << "Encode decode test passed!" << std::endl;

	CUDA_CHECK_LAST();

}

void test_fnt_performance() {

	// fnt() performance with memory already prepare in device

	using namespace std;

	const unsigned N_test = 1024 * 1024 / 64;
	//const unsigned N_test = 1; // use when need profile one..
	unsigned log_n = 16, n = 1 << log_n;
	unsigned size_n = n * sizeof(unsigned);
	vector<vector<unsigned>> a(N_test, vector<unsigned>(n));
	hipStream_t stream[N_test];
	vector<unsigned*> d_a(N_test), d_b(N_test);

	for (unsigned tt = 0; tt < N_test; tt++) {
		for (unsigned i = 0; i < n; i++)
			a[tt][i] = rand() % (MOD - 1);
		CUDA_CHECK(hipStreamCreate(&stream[tt]));
		CUDA_CHECK(hipMallocAsync(&d_a[tt], size_n, stream[tt]));
		CUDA_CHECK(hipMallocAsync(&d_b[tt], size_n, stream[tt]));
		CUDA_CHECK(hipMemcpyAsync(d_a[tt], a[tt].data(), size_n, hipMemcpyHostToDevice, stream[tt]));
	}

	CUDA_CHECK(hipDeviceSynchronize());

	cout << "FNT test start" << endl;

	auto start = chrono::high_resolution_clock::now();
	unsigned wpt = n >> (LOG_THREAD_PER_OP + 1);

	for (unsigned tt = 0; tt < N_test; tt++) {
		//hipStreamCreate(&stream[tt]);
		//hipMallocAsync(&d_a[tt], size_n, stream[tt]);
		//hipMallocAsync(&d_b[tt], size_n, stream[tt]);
		//hipMemcpyAsync(d_a[tt], a[tt].data(), size_n, hipMemcpyHostToDevice, stream[tt]);
		fnt CUDA_KERNEL(1, THREAD_PER_OP, NULL, stream[tt])(d_a[tt], d_b[tt], log_n, log_n, 0, d_N_pos, d_root_layer_pow, d_inv, wpt);
		CUDA_CHECK_LAST();
		//hipFreeAsync(d_a[tt], stream[tt]);
		//hipFreeAsync(d_b[tt], stream[tt]);
		//hipStreamDestroy(stream[tt]);
	}

	CUDA_CHECK(hipDeviceSynchronize());
	auto stop = chrono::high_resolution_clock::now();
	auto duration = chrono::duration_cast<chrono::milliseconds>(stop - start).count();

	cout << "FNT " << N_test << " chunks in " << duration << "ms" << endl;

	for (unsigned tt = 0; tt < N_test; tt++) {
		CUDA_CHECK(hipFreeAsync(d_a[tt], stream[tt]));
		CUDA_CHECK(hipFreeAsync(d_b[tt], stream[tt]));
		CUDA_CHECK(hipStreamDestroy(stream[tt]));
	}

}

void test_encode_decode_performance() {

	// test encode(), decode() performance full flow (without prepare memory in device)

	using namespace std;
	srand(time(NULL));

	const unsigned N_test = 128 * 1024 / 64;
	//const unsigned N_test = 1; // use when need profile one..
	const long long symbol_bytes = 2;
	const double size_test_gb = 1.0 * symbol_bytes * NUM_OF_NEED_SYMBOL * N_test / (1024 * 1024 * 1024);

	vector<vector<unsigned>> a(N_test, vector<unsigned>(NUM_OF_NEED_SYMBOL));
	vector<vector<unsigned>> b(N_test, vector<unsigned>(NUM_OF_NEED_SYMBOL << 1));
	vector<vector<unsigned>> c(N_test, vector<unsigned>(NUM_OF_NEED_SYMBOL));

	for (unsigned tt = 0; tt < N_test; tt++)
		for (unsigned i = 0; i < NUM_OF_NEED_SYMBOL; i++)
			a[tt][i] = rand() % (MOD - 1); // 2 bytes

	CUDA_CHECK(hipDeviceSynchronize());

	cout << "Encode performance test start" << endl;

	auto start1 = chrono::high_resolution_clock::now();

	for (unsigned tt = 0; tt < N_test; tt++) {
		h_encode(a[tt].data(), b[tt].data());
		CUDA_CHECK_LAST();
	}

	CUDA_CHECK(hipDeviceSynchronize());
	auto stop1 = chrono::high_resolution_clock::now();
	auto duration1 = chrono::duration_cast<chrono::milliseconds>(stop1 - start1).count();

	cout << "Encode " << N_test << " 64kb chunks in " << duration1 << "ms" << endl;
	cout << "Encode " << (1.0 * size_test_gb) / (1.0 * duration1 / 1000.0) << " GB/s" << endl;

	vector<vector<unsigned>> x(N_test, vector<unsigned>(NUM_OF_NEED_SYMBOL));
	vector<vector<unsigned>> y(N_test, vector<unsigned>(NUM_OF_NEED_SYMBOL));

	for (unsigned tt = 0; tt < N_test; tt++) {
		for (unsigned i = 0; i < NUM_OF_NEED_PACKET; i++) {
			unsigned stx = i * SYMBOL_PER_PACKET;
			for (unsigned j = 0; j < SEG_PER_PACKET; j++) {
				x[tt][stx + j] = stx + j;
				x[tt][stx + j + SEG_PER_PACKET] = stx + j + SEG_DIFF;
				y[tt][stx + j] = b[tt][stx + j];
				y[tt][stx + j + SEG_PER_PACKET] = b[tt][stx + j + SEG_DIFF];
			}
		}
	}

	CUDA_CHECK(hipDeviceSynchronize());

	cout << "Decode performance test start" << endl;

	auto start2 = chrono::high_resolution_clock::now();

	for (unsigned tt = 0; tt < N_test; tt++) {
		h_decode(x[tt].data(), y[tt].data(), c[tt].data());
		CUDA_CHECK_LAST();
	}

	CUDA_CHECK(hipDeviceSynchronize());
	auto stop2 = chrono::high_resolution_clock::now();
	auto duration2 = chrono::duration_cast<chrono::milliseconds>(stop2 - start2).count();

	cout << "Decode " << N_test << " 64kb chunks in " << duration2 << "ms" << endl;
	cout << "Decode " << (1.0 * size_test_gb) / (1.0 * duration2 / 1000.0) << " GB/s" << endl;

	for (unsigned tt = 0; tt < N_test; tt++) {
		for (unsigned i = 0; i < c[tt].size(); i++)
			assert(a[tt][i] == c[tt][i]);
	}

}
