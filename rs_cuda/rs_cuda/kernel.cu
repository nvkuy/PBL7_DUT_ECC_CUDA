﻿#ifdef __HIPCC__
#define CUDA_KERNEL(grid, block, sh_mem, stream) <<< grid, block, sh_mem, stream >>>
#define CUDA_SYNCTHREADS() __syncthreads()
#else
#define CUDA_KERNEL(grid, block, sh_mem, stream)
#define CUDA_SYNCTHREADS()
#endif

#include <hip/hip_runtime.h>


#include <hip/hip_runtime_api.h>

#include <iostream>
#include <time.h>
#include <vector>
#include <cassert>
#include <algorithm>
#include <random>
#include <chrono>
#include <queue>
#include <mutex>
#include <condition_variable>

#define CUDA_CHECK(val) check((val), #val, __FILE__, __LINE__)
inline void check(hipError_t err, const char* const func, const char* const file, const int line)
{
	if (err != hipSuccess)
	{
		std::cerr << "CUDA Runtime Error at: " << file << ":" << line << std::endl;
		std::cerr << hipGetErrorString(err) << " " << func << std::endl;
		std::exit(EXIT_FAILURE);
	}
}

#define CUDA_CHECK_LAST() check_last(__FILE__, __LINE__)
inline void check_last(const char* const file, const int line)
{
	hipError_t const err{ hipPeekAtLastError() };
	if (err != hipSuccess)
	{
		std::cerr << "CUDA Runtime Error at: " << file << ":" << line << std::endl;
		std::cerr << hipGetErrorString(err) << std::endl;
		std::exit(EXIT_FAILURE);
	}
}

const unsigned MOD = 65537;
const unsigned SPECIAL = MOD - 1;
const unsigned ROOT = 3;
const unsigned ROOT_INV = 21846;
const unsigned MAX_LOG = 16;

const unsigned LOG_DATA = 10;
const unsigned LOG_SYMBOL = LOG_DATA - 1;
const unsigned LOG_SEG = LOG_SYMBOL - 1;
const unsigned SYMBOL_PER_PACKET = 1 << LOG_SYMBOL;
const unsigned NUM_OF_PACKET = 1 << (MAX_LOG - LOG_SYMBOL);
const unsigned NUM_OF_NEED_PACKET = NUM_OF_PACKET >> 1;
const unsigned SEG_PER_PACKET = 1 << LOG_SEG;
const unsigned SEG_DIFF = 1 << (MAX_LOG - 1);
const unsigned NUM_OF_NEED_SYMBOL = 1 << (MAX_LOG - 1);

const unsigned LEN_ROOT_LAYER_POW = (1 << MAX_LOG) - 1;
const unsigned LEN_ROOT_LAYER_POW_2 = LEN_ROOT_LAYER_POW << 1;
const unsigned LEN_N_POS = ((1 << (MAX_LOG + 1)) - 1);
const unsigned LEN_PACKET_PRODUCT = NUM_OF_PACKET * (SYMBOL_PER_PACKET << 1);
const unsigned LEN_ONE_PACKET_PRODUCT = 1 << (LOG_SYMBOL + 1);

const unsigned LEN_SMALL = NUM_OF_NEED_SYMBOL;
const unsigned LEN_LARGE = LEN_SMALL << 1;

const unsigned SIZE_SMALL = LEN_SMALL * sizeof(unsigned);
const unsigned SIZE_LARGE = LEN_LARGE * sizeof(unsigned);
const unsigned SIZE_ONE_PACKET_PRODUCT = LEN_ONE_PACKET_PRODUCT * sizeof(unsigned);

const unsigned LOG_LEN_ENCODE_P = MAX_LOG - 1;
const unsigned LOG_LEN_ENCODE_Y = MAX_LOG;
const unsigned LOG_LEN_DECODE_X = MAX_LOG - 1;
const unsigned LOG_LEN_DECODE_Y = MAX_LOG - 1;
const unsigned LOG_LEN_DECODE_P = MAX_LOG - 1;

const unsigned LEN_ENCODE_P = 1 << LOG_LEN_ENCODE_P;
const unsigned LEN_ENCODE_Y = 1 << LOG_LEN_ENCODE_Y;
const unsigned LEN_DECODE_X = 1 << LOG_LEN_DECODE_X;
const unsigned LEN_DECODE_Y = 1 << LOG_LEN_DECODE_Y;
const unsigned LEN_DECODE_P = 1 << LOG_LEN_DECODE_P;

const unsigned SIZE_ENCODE_P = LEN_ENCODE_P * sizeof(unsigned);
const unsigned SIZE_ENCODE_Y = LEN_ENCODE_Y * sizeof(unsigned);
const unsigned SIZE_DECODE_X = LEN_DECODE_X * sizeof(unsigned);
const unsigned SIZE_DECODE_Y = LEN_DECODE_Y * sizeof(unsigned);
const unsigned SIZE_DECODE_P = LEN_DECODE_P * sizeof(unsigned);

const unsigned MAX_ACTIVE_ENCODE = 256;
const unsigned MAX_ACTIVE_DECODE = 256;
const unsigned MAX_ENCODE_LAUNCH_CNT = 16;
const unsigned MAX_DECODE_LAUNCH_CNT = 128;

const unsigned LOG_ALGO_LOW_WPT = 3;
const unsigned LOG_ALGO_MED_WPT = 2;
const unsigned LOG_ALGO_HIGH_WPT = 2;
const unsigned LOG_ALGO_EXTR_WPT = 0;

const unsigned LOG_LEN_WARP = 5;
const unsigned LEN_WARP = 1 << LOG_LEN_WARP;
const unsigned ALGO_N_2_CUTOFF = 64;

const unsigned ALGO_LOW_WPT = 1 << LOG_ALGO_LOW_WPT;
const unsigned ALGO_MED_WPT = 1 << LOG_ALGO_MED_WPT;
const unsigned ALGO_HIGH_WPT = 1 << LOG_ALGO_HIGH_WPT; 
const unsigned ALGO_EXTR_WPT = 1 << LOG_ALGO_EXTR_WPT; 

const size_t SIZE_ENCODE_P_SLOT = sizeof(unsigned) * LEN_ENCODE_P * MAX_ACTIVE_ENCODE;
const size_t SIZE_ENCODE_Y_SLOT = sizeof(unsigned) * LEN_ENCODE_Y * MAX_ACTIVE_ENCODE;

const size_t SIZE_DECODE_X_SLOT = sizeof(unsigned) * LEN_DECODE_X * MAX_ACTIVE_DECODE;
const size_t SIZE_DECODE_Y_SLOT = sizeof(unsigned) * LEN_DECODE_Y * MAX_ACTIVE_DECODE;
//const size_t SIZE_DECODE_P_SLOT = sizeof(unsigned) * LEN_DECODE_P * MAX_ACTIVE_DECODE;
const size_t SIZE_DECODE_T1_SLOT = sizeof(unsigned) * LEN_LARGE * MAX_ACTIVE_DECODE;
const size_t SIZE_DECODE_T2_SLOT = sizeof(unsigned) * LEN_LARGE * MAX_ACTIVE_DECODE;
const size_t SIZE_DECODE_AX_SLOT = sizeof(unsigned) * LEN_LARGE * MAX_ACTIVE_DECODE;
const size_t SIZE_DECODE_DAX_SLOT = sizeof(unsigned) * LEN_SMALL * MAX_ACTIVE_DECODE;
const size_t SIZE_DECODE_VDAX_SLOT = sizeof(unsigned) * LEN_LARGE * MAX_ACTIVE_DECODE;
const size_t SIZE_DECODE_N1_SLOT = sizeof(unsigned) * LEN_SMALL * MAX_ACTIVE_DECODE;
const size_t SIZE_DECODE_N2_SLOT = sizeof(unsigned) * LEN_LARGE * MAX_ACTIVE_DECODE;
const size_t SIZE_DECODE_N3_SLOT = sizeof(unsigned) * LEN_SMALL * MAX_ACTIVE_DECODE;

unsigned** h_encode_p_slot;
unsigned** h_encode_y_slot;
unsigned** h_decode_x_slot;
unsigned** h_decode_y_slot;
unsigned** h_decode_p_slot;

unsigned* d_encode_p_slot;
unsigned* d_encode_y_slot;
unsigned* d_decode_x_slot;
unsigned* d_decode_y_slot;
//unsigned* d_decode_p_slot;
unsigned* d_decode_t1_slot;
unsigned* d_decode_t2_slot;
unsigned* d_decode_ax_slot;
unsigned* d_decode_dax_slot;
unsigned* d_decode_vdax_slot;
unsigned* d_decode_n1_slot;
unsigned* d_decode_n2_slot;
unsigned* d_decode_n3_slot;

unsigned* d_N_pos;
unsigned* d_root_pow;
unsigned* d_root_inv_pow;
unsigned* d_inv;
unsigned* d_root_layer_pow;
unsigned* d_packet_product;

struct CB_DATA {
	unsigned slot_id;
	unsigned* dst;
	unsigned* src;
	size_t size_res;
	std::queue<unsigned>& slot; 
	std::mutex& mt; 
	std::condition_variable& cv;
};

std::queue<unsigned> encode_slot, decode_slot;
std::mutex mt_encode_slot, mt_decode_slot;
std::condition_variable cv_encode_slot, cv_decode_slot;

inline unsigned pop_slot(std::queue<unsigned> &slot, std::mutex &mt, std::condition_variable &cv) {
	std::unique_lock<std::mutex> lock(mt);
	cv.wait(lock, [&] { return !slot.empty(); });
	unsigned id = slot.front();
	slot.pop();
	return id;
}

inline void push_slot(unsigned id, std::queue<unsigned>& slot, std::mutex& mt, std::condition_variable& cv) {
	{
		std::lock_guard<std::mutex> lock(mt);
		slot.push(id);
	}
	cv.notify_one();
}

void init_batch_slot() {

	for (unsigned i = 0; i < MAX_ACTIVE_ENCODE; i++)
		push_slot(i, encode_slot, mt_encode_slot, cv_encode_slot);

	for (unsigned i = 0; i < MAX_ACTIVE_DECODE; i++)
		push_slot(i, decode_slot, mt_decode_slot, cv_decode_slot);

}

void CUDART_CB h_end_batch_slot(void* data) {

	CB_DATA* dat = static_cast<CB_DATA*>(data);

	memcpy(dat->dst, dat->src, dat->size_res);
	push_slot(dat->slot_id, dat->slot, dat->mt, dat->cv);

	delete dat;
}

__host__ __device__ __forceinline__ inline void build_launch_param(unsigned log_n, unsigned& n_th, unsigned& n_bl) {
	if (log_n <= LOG_LEN_WARP) {
		n_th = 1 << log_n;
		n_bl = 1;
	}
	else {
		unsigned c_l2_sqrt_n = (log_n >> 1) + (log_n & 1);
		n_th = 1 << c_l2_sqrt_n;
		n_bl = 1 << (log_n - c_l2_sqrt_n);
	}
}

__host__ __device__ __forceinline__ inline unsigned mul_mod(unsigned a, unsigned b)
{
	if (a == SPECIAL && b == SPECIAL)
		return 1; // overflow
	return (a * b) % MOD;
}

__device__ __forceinline__ inline unsigned div_mod(unsigned a, unsigned b,
	unsigned* d_inv)
{
	return mul_mod(a, d_inv[b]);
}

__host__ __device__ __forceinline__ inline unsigned add_mod(unsigned a, unsigned b)
{
	return (a + b) % MOD;
}

__host__ __device__ __forceinline__ inline unsigned sub_mod(unsigned a, unsigned b)
{
	return (a - b + MOD) % MOD;
}

__host__ __device__ __forceinline__ inline unsigned pow_mod(unsigned a, unsigned b)
{
	unsigned res = 1;
	while (b > 0)
	{
		if (b & 1)
			res = mul_mod(res, a);
		a = mul_mod(a, a);
		b >>= 1;
	}
	return res;
}

__global__ void g_pre_fnt(unsigned* a, unsigned* b, unsigned st, unsigned* d_N_pos)
{

	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned id_l = id * ALGO_HIGH_WPT, id_r = id_l + ALGO_HIGH_WPT;
	for (unsigned k = id_l; k < id_r; k++)
		b[d_N_pos[st + k]] = a[k];

}

__global__ void g_end_fnt(unsigned* b, unsigned n, unsigned* d_inv)
{

	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned id_l = id * ALGO_HIGH_WPT, id_r = id_l + ALGO_HIGH_WPT;
	for (unsigned k = id_l; k < id_r; k++) {
		b[k << 1] = div_mod(b[k << 1], n, d_inv);
		b[(k << 1) | 1] = div_mod(b[(k << 1) | 1], n, d_inv);
	}
}

__global__ void g_fnt_i(unsigned* b, unsigned i, bool inv,
	unsigned* d_root_layer_pow)
{

	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned id_l = id * ALGO_HIGH_WPT, id_r = id_l + ALGO_HIGH_WPT;

	unsigned haft_len = 1 << i;
	for (unsigned k = id_l; k < id_r; k++) {
		unsigned bl_st = ((k >> i) << (i + 1)), th_id = (k & (haft_len - 1));
		unsigned pos = bl_st + th_id;
		unsigned u = b[pos];
		unsigned v = mul_mod(b[pos + haft_len], d_root_layer_pow[(LEN_ROOT_LAYER_POW * inv) + haft_len - 1 + th_id]);
		b[pos] = add_mod(u, v);
		b[pos + haft_len] = sub_mod(u, v);
	}

}

__host__ __forceinline__ __device__ inline void fnt(unsigned* a, unsigned* b, unsigned log_na, unsigned log_nb, unsigned opt,
	unsigned* d_N_pos, unsigned* d_root_layer_pow, unsigned* d_inv, hipStream_t stream)
{

	/*
	opt 2 bit: x1 x2
	 - x1: w_n or 1/w_n
	 - x2: need result * 1/n
	*/

	// size_b >= size_a;
	// need memset *b before use unless size_a == size_b

	unsigned nb = 1 << log_nb, wp = (opt & 2) >> 1;
	unsigned n_bl, n_th;

	build_launch_param(log_na - LOG_ALGO_HIGH_WPT, n_th, n_bl);
	g_pre_fnt CUDA_KERNEL(n_bl, n_th, NULL, stream)(a, b, nb - 1, d_N_pos);

	build_launch_param(log_nb - 1 - LOG_ALGO_HIGH_WPT, n_th, n_bl);
	for (unsigned i = 0; i < log_nb; i++)
		g_fnt_i CUDA_KERNEL(n_bl, n_th, NULL, stream)(b, i, wp, d_root_layer_pow);

	if (opt & 1)
		g_end_fnt CUDA_KERNEL(n_bl, n_th, NULL, stream)(b, nb, d_inv);
}

__global__ void g_vector_mul_i(unsigned* a, unsigned* b, unsigned* c)
{

	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned id_l = id * ALGO_LOW_WPT, id_r = id_l + ALGO_LOW_WPT;
	for (unsigned k = id_l; k < id_r; k++)
		c[k] = mul_mod(a[k], b[k]);


}

__global__ void g_fill(unsigned* a, unsigned val) {

	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned id_l = id * ALGO_LOW_WPT, id_r = id_l + ALGO_LOW_WPT;
	for (unsigned k = id_l; k < id_r; k++)
		a[k] = val;

}

__forceinline__ __device__ void d_poly_mul(unsigned* a, unsigned* b, unsigned* t1, unsigned* t2, unsigned* c, unsigned log_n,
	unsigned* d_N_pos, unsigned* d_root_layer_pow, unsigned* d_inv)
{

	// 2 ^ log_n == size_a && size_a == size_b
	// *c == *a && *a + na == *b (allow)

	unsigned na = 1 << log_n, nc = na << 1, size_nc = nc * sizeof(unsigned);

	if (na <= ALGO_N_2_CUTOFF)
	{
		for (unsigned i = 0; i < na; i++)
		{
			t1[i] = a[i];
			t2[i] = b[i];
		}
		memset(c, 0, size_nc);
		for (unsigned i = 0; i < na; i++)
			for (unsigned j = 0; j < na; j++)
				c[i + j] = add_mod(c[i + j], mul_mod(t1[i], t2[j]));
	}
	else
	{

		hipStream_t stream;
		hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);

		unsigned n_bl, n_th;
		build_launch_param(log_n + 1 - LOG_ALGO_LOW_WPT, n_th, n_bl);
		g_fill CUDA_KERNEL(n_bl, n_th, NULL, stream) (t1, 0);
		g_fill CUDA_KERNEL(n_bl, n_th, NULL, stream) (t2, 0);

		fnt(a, t1, log_n, log_n + 1, 0, d_N_pos, d_root_layer_pow, d_inv, stream);
		fnt(b, t2, log_n, log_n + 1, 0, d_N_pos, d_root_layer_pow, d_inv, stream);

		g_vector_mul_i CUDA_KERNEL(n_bl, n_th, NULL, stream)(t1, t2, t1);

		fnt(t1, c, log_n + 1, log_n + 1, 3, d_N_pos, d_root_layer_pow, d_inv, stream);

		hipStreamDestroy(stream);

	}
}

inline void h_poly_mul(unsigned* a, unsigned* b, unsigned* t1, unsigned* t2, unsigned* c, unsigned log_n,
	unsigned* d_N_pos, unsigned* d_root_layer_pow, unsigned* d_inv, hipStream_t stream) {

	// only use with large poly

	unsigned nc = 1 << (log_n + 1), size_nc = nc * sizeof(unsigned);

	unsigned n_bl, n_th;
	build_launch_param(log_n + 1 - LOG_ALGO_LOW_WPT, n_th, n_bl);
	g_fill CUDA_KERNEL(n_bl, n_th, NULL, stream) (t1, 0);
	g_fill CUDA_KERNEL(n_bl, n_th, NULL, stream) (t2, 0);

	fnt(a, t1, log_n, log_n + 1, 0, d_N_pos, d_root_layer_pow, d_inv, stream);
	fnt(b, t2, log_n, log_n + 1, 0, d_N_pos, d_root_layer_pow, d_inv, stream);

	g_vector_mul_i CUDA_KERNEL(n_bl, n_th, NULL, stream)(t1, t2, t1);

	fnt(t1, c, log_n + 1, log_n + 1, 3, d_N_pos, d_root_layer_pow, d_inv, stream);

}

__global__ void g_poly_deriv(unsigned* ax, unsigned* dax)
{

	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned id_l = id * ALGO_MED_WPT, id_r = id_l + ALGO_MED_WPT;
	for (unsigned k = id_l; k < id_r; k++)
		dax[k] = mul_mod(ax[k + 1], k + 1);
}

__global__ void g_build_product_i(unsigned* p, unsigned* t1, unsigned* t2, unsigned i,
	unsigned* d_N_pos, unsigned* d_root_layer_pow, unsigned* d_inv)
{

	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned len = 1 << i;

	unsigned id_l = id * ALGO_EXTR_WPT, id_r = id_l + ALGO_EXTR_WPT;
	for (unsigned k = id_l; k < id_r; k++) {
		unsigned st = k << (i + 1);
		d_poly_mul(p + st, p + st + len, t1 + st, t2 + st, p + st, i, d_N_pos, d_root_layer_pow, d_inv);
	}

}

inline void h_build_product(unsigned* p, unsigned* t1, unsigned* t2, unsigned log_n1, unsigned log_n2, hipStream_t stream)
{

	// p, t1, t2 in device

	for (unsigned i = log_n1; i < log_n2; i++)
	{
		unsigned n_th, n_bl;
		build_launch_param(log_n2 - i - 1 - LOG_ALGO_EXTR_WPT, n_th, n_bl);
		g_build_product_i CUDA_KERNEL(n_bl, n_th, NULL, stream)(p, t1, t2, i, d_N_pos, d_root_layer_pow, d_inv);
		CUDA_CHECK_LAST();
	}
}

inline void h_build_ax(unsigned* x, unsigned* p, unsigned* t1, unsigned* t2, hipStream_t stream)
{

	// p, t1, t2 in device
	// x in host

	for (unsigned i = 0; i < NUM_OF_NEED_PACKET; i++)
	{
		unsigned st_p1 = i << (LOG_SYMBOL + 1), st_p2 = x[i << LOG_SYMBOL] << 2;
		CUDA_CHECK(hipMemcpyAsync(p + st_p1, d_packet_product + st_p2, SIZE_ONE_PACKET_PRODUCT, hipMemcpyDeviceToDevice, stream));
	}
	h_build_product(p, t1, t2, LOG_SYMBOL + 1, MAX_LOG, stream);
}

__global__ void g_build_n1(unsigned* n1, unsigned* vdax, unsigned* x, unsigned* y,
	unsigned* d_inv)
{

	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned id_l = id * ALGO_MED_WPT, id_r = id_l + ALGO_MED_WPT;
	for (unsigned k = id_l; k < id_r; k++)
		n1[k] = div_mod(y[k], vdax[x[k]], d_inv);

}

__global__ void g_build_n2(unsigned* n2, unsigned* n1, unsigned* x) {

	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned id_l = id * ALGO_MED_WPT, id_r = id_l + ALGO_MED_WPT;
	for (unsigned k = id_l; k < id_r; k++)
		n2[x[k]] = n1[k];

}

__global__ void g_build_n3(unsigned* n3, unsigned* p_n3) {

	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned id_l = id * ALGO_MED_WPT, id_r = id_l + ALGO_MED_WPT;
	for (unsigned k = id_l; k < id_r; k++)
		n3[k] = sub_mod(0, p_n3[k + 1]);

}

inline void h_build_px(unsigned* p, unsigned* ax, unsigned* n3, unsigned* t1, unsigned* t2,
	unsigned* d_N_pos, unsigned* d_root_layer_pow, unsigned* d_inv, hipStream_t stream) {

	h_poly_mul(ax, n3, t1, t2, p, MAX_LOG - 1, d_N_pos, d_root_layer_pow, d_inv, stream);
	CUDA_CHECK_LAST();

}

void h_encode(unsigned* p, unsigned* y)
{

	unsigned slot_id = pop_slot(encode_slot, mt_encode_slot, cv_encode_slot);

	unsigned* sl_p = h_encode_p_slot[slot_id];
	unsigned* sl_y = h_encode_y_slot[slot_id];

	memcpy(sl_p, p, SIZE_ENCODE_P);

	unsigned* d_p = d_encode_p_slot + 1LL * slot_id * LEN_ENCODE_P;
	unsigned* d_y = d_encode_y_slot + 1LL * slot_id * LEN_ENCODE_Y;

	unsigned n_th, n_bl;
	build_launch_param(LOG_LEN_ENCODE_Y - LOG_ALGO_LOW_WPT, n_th, n_bl);

	CB_DATA* data = new CB_DATA{ slot_id, y, sl_y, SIZE_ENCODE_Y, encode_slot, mt_encode_slot, cv_encode_slot };

	hipStream_t stream;
	CUDA_CHECK(hipStreamCreate(&stream));

	CUDA_CHECK(hipMemcpyAsync(d_p, sl_p, SIZE_ENCODE_P, hipMemcpyHostToDevice, stream));
	
	g_fill CUDA_KERNEL(n_bl, n_th, NULL, stream)(d_y, 0);

	fnt(d_p, d_y, MAX_LOG - 1, MAX_LOG, 0, d_N_pos, d_root_layer_pow, d_inv, stream);
	CUDA_CHECK_LAST();

	CUDA_CHECK(hipMemcpyAsync(sl_y, d_y, SIZE_ENCODE_Y, hipMemcpyDeviceToHost, stream));
	
	CUDA_CHECK(hipLaunchHostFunc(stream, h_end_batch_slot, data));

	CUDA_CHECK(hipStreamDestroy(stream));

}

void h_decode(unsigned* x, unsigned* y, unsigned* p)
{

	unsigned slot_id = pop_slot(decode_slot, mt_decode_slot, cv_decode_slot);

	unsigned* sl_x = h_decode_x_slot[slot_id];
	unsigned* sl_y = h_decode_y_slot[slot_id];
	unsigned* sl_p = h_decode_p_slot[slot_id];

	memcpy(sl_x, x, SIZE_DECODE_X);
	memcpy(sl_y, y, SIZE_DECODE_Y);

	unsigned* d_x = d_decode_x_slot + 1LL * slot_id * LEN_DECODE_X;
	unsigned* d_y = d_decode_y_slot + 1LL * slot_id * LEN_DECODE_Y;
	unsigned* d_t1 = d_decode_t1_slot + 1LL * slot_id * LEN_LARGE;
	unsigned* d_t2 = d_decode_t2_slot + 1LL * slot_id * LEN_LARGE;
	unsigned* d_ax = d_decode_ax_slot + 1LL * slot_id * LEN_LARGE;
	unsigned* d_dax = d_decode_dax_slot + 1LL * slot_id * LEN_SMALL;
	unsigned* d_vdax = d_decode_vdax_slot + 1LL * slot_id * LEN_LARGE;
	unsigned* d_n1 = d_decode_n1_slot + 1LL * slot_id * LEN_SMALL;
	unsigned* d_n2 = d_decode_n2_slot + 1LL * slot_id * LEN_LARGE;
	unsigned* d_n3 = d_decode_n3_slot + 1LL * slot_id * LEN_SMALL;

	unsigned n_th1, n_bl1, n_th2, n_bl2;
	build_launch_param(MAX_LOG - LOG_ALGO_MED_WPT - 1, n_th1, n_bl1);
	build_launch_param(MAX_LOG - LOG_ALGO_LOW_WPT, n_th2, n_bl2);

	CB_DATA* data = new CB_DATA{ slot_id, p, sl_p, SIZE_DECODE_P, decode_slot, mt_decode_slot, cv_decode_slot };

	hipStream_t stream;
	CUDA_CHECK(hipStreamCreate(&stream));

	CUDA_CHECK(hipMemcpyAsync(d_x, sl_x, SIZE_DECODE_X, hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(d_y, sl_y, SIZE_DECODE_Y, hipMemcpyHostToDevice, stream));

	g_fill CUDA_KERNEL(n_bl2, n_th2, NULL, stream)(d_vdax, 0);
	g_fill CUDA_KERNEL(n_bl2, n_th2, NULL, stream)(d_n2, 0);

	h_build_ax(x, d_ax, d_t1, d_t2, stream);

	g_poly_deriv CUDA_KERNEL(n_bl1, n_th1, NULL, stream)(d_ax, d_dax);
	CUDA_CHECK_LAST();
	
	fnt(d_dax, d_vdax, MAX_LOG - 1, MAX_LOG, 0, d_N_pos, d_root_layer_pow, d_inv, stream);
	CUDA_CHECK_LAST();
	
	g_build_n1 CUDA_KERNEL(n_bl1, n_th1, NULL, stream)(d_n1, d_vdax, d_x, d_y, d_inv);
	CUDA_CHECK_LAST();
	
	g_build_n2 CUDA_KERNEL(n_bl1, n_th1, NULL, stream)(d_n2, d_n1, d_x);
	CUDA_CHECK_LAST();
	
	fnt(d_n2, d_t2, MAX_LOG, MAX_LOG, 2, d_N_pos, d_root_layer_pow, d_inv, stream);
	CUDA_CHECK_LAST();
	g_build_n3 CUDA_KERNEL(n_bl1, n_th1, NULL, stream)(d_n3, d_t2);
	CUDA_CHECK_LAST();
	
	h_build_px(d_n2, d_ax, d_n3, d_t1, d_t2, d_N_pos, d_root_layer_pow, d_inv, stream);
	CUDA_CHECK(hipMemcpyAsync(sl_p, d_n2, SIZE_SMALL, hipMemcpyDeviceToHost, stream));

	CUDA_CHECK(hipLaunchHostFunc(stream, h_end_batch_slot, data));

	CUDA_CHECK(hipStreamDestroy(stream));

}

void init()
{
	// offline process

	CUDA_CHECK(hipDeviceSynchronize());
	CUDA_CHECK(hipDeviceReset());
	CUDA_CHECK(hipDeviceSetLimit(cudaLimitDevRuntimePendingLaunchCount, 
		std::max(MAX_ACTIVE_DECODE * MAX_DECODE_LAUNCH_CNT, MAX_ACTIVE_ENCODE * MAX_ENCODE_LAUNCH_CNT)));

	h_encode_p_slot = (unsigned**)malloc(MAX_ACTIVE_ENCODE * sizeof(unsigned*));
	h_encode_y_slot = (unsigned**)malloc(MAX_ACTIVE_ENCODE * sizeof(unsigned*));
	h_decode_x_slot = (unsigned**)malloc(MAX_ACTIVE_DECODE * sizeof(unsigned*));
	h_decode_y_slot = (unsigned**)malloc(MAX_ACTIVE_DECODE * sizeof(unsigned*));
	h_decode_p_slot = (unsigned**)malloc(MAX_ACTIVE_DECODE * sizeof(unsigned*));

	for (unsigned i = 0; i < MAX_ACTIVE_ENCODE; i++) {
		CUDA_CHECK(hipHostMalloc(&(h_encode_p_slot[i]), SIZE_ENCODE_P, hipHostMallocDefault));
		CUDA_CHECK(hipHostMalloc(&(h_encode_y_slot[i]), SIZE_ENCODE_Y, hipHostMallocDefault));
	}

	for (unsigned i = 0; i < MAX_ACTIVE_DECODE; i++) {
		CUDA_CHECK(hipHostMalloc(&(h_decode_x_slot[i]), SIZE_DECODE_X, hipHostMallocDefault));
		CUDA_CHECK(hipHostMalloc(&(h_decode_y_slot[i]), SIZE_DECODE_Y, hipHostMallocDefault));
		CUDA_CHECK(hipHostMalloc(&(h_decode_p_slot[i]), SIZE_DECODE_P, hipHostMallocDefault));
	}

	CUDA_CHECK(hipMalloc(&d_encode_p_slot, SIZE_ENCODE_P_SLOT));
	CUDA_CHECK(hipMalloc(&d_encode_y_slot, SIZE_ENCODE_Y_SLOT));

	CUDA_CHECK(hipMalloc(&d_decode_x_slot, SIZE_DECODE_X_SLOT));
	CUDA_CHECK(hipMalloc(&d_decode_y_slot, SIZE_DECODE_Y_SLOT));
	//CUDA_CHECK(cudaMalloc(&d_decode_p_slot, SIZE_DECODE_P_SLOT));
	CUDA_CHECK(hipMalloc(&d_decode_t1_slot, SIZE_DECODE_T1_SLOT));
	CUDA_CHECK(hipMalloc(&d_decode_t2_slot, SIZE_DECODE_T2_SLOT));
	CUDA_CHECK(hipMalloc(&d_decode_ax_slot, SIZE_DECODE_AX_SLOT));
	CUDA_CHECK(hipMalloc(&d_decode_dax_slot, SIZE_DECODE_DAX_SLOT));
	CUDA_CHECK(hipMalloc(&d_decode_vdax_slot, SIZE_DECODE_VDAX_SLOT));
	CUDA_CHECK(hipMalloc(&d_decode_n1_slot, SIZE_DECODE_N1_SLOT));
	CUDA_CHECK(hipMalloc(&d_decode_n2_slot, SIZE_DECODE_N2_SLOT));
	CUDA_CHECK(hipMalloc(&d_decode_n3_slot, SIZE_DECODE_N3_SLOT));

	init_batch_slot();

	unsigned size_N_pos = LEN_N_POS * sizeof(unsigned);
	unsigned* N_pos = (unsigned*)malloc(size_N_pos);
	CUDA_CHECK(hipMalloc(&d_N_pos, size_N_pos));

	for (unsigned i = 1; i <= MAX_LOG; i++)
	{
		unsigned n = 1 << i, st = n - 1;
		for (unsigned j = 0; j < n; j++)
			N_pos[st + j] = j;
	}

	for (unsigned i = 1; i <= MAX_LOG; i++)
	{
		unsigned n = 1 << i, st = n - 1;
		for (unsigned j = 0; j < n; j++)
		{
			unsigned rev_num = 0;
			for (unsigned k = 0; k < i; k++)
			{
				if (j & (1 << k))
					rev_num |= (1 << (i - 1 - k));
			}
			if (j < rev_num)
				std::swap(N_pos[st + j], N_pos[st + rev_num]);
		}
	}

	CUDA_CHECK(hipMemcpy(d_N_pos, N_pos, size_N_pos, hipMemcpyHostToDevice));
	free(N_pos);

	unsigned size_root = MOD * sizeof(unsigned);
	unsigned* root_pow = (unsigned*)malloc(size_root);
	unsigned* root_inv_pow = (unsigned*)malloc(size_root);
	unsigned* inv = (unsigned*)malloc(size_root);
	CUDA_CHECK(hipMalloc(&d_root_pow, size_root));
	CUDA_CHECK(hipMalloc(&d_root_inv_pow, size_root));
	CUDA_CHECK(hipMalloc(&d_inv, size_root));

	root_pow[0] = 1, root_inv_pow[0] = 1, inv[0] = 0;
	for (unsigned i = 1; i < MOD; i++)
	{
		root_pow[i] = mul_mod(root_pow[i - 1], ROOT);
		root_inv_pow[i] = mul_mod(root_inv_pow[i - 1], ROOT_INV);
		inv[i] = pow_mod(i, MOD - 2);
	}

	CUDA_CHECK(hipMemcpy(d_root_pow, root_pow, size_root, hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(d_root_inv_pow, root_inv_pow, size_root, hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(d_inv, inv, size_root, hipMemcpyHostToDevice));

	unsigned size_root_layer_pow = LEN_ROOT_LAYER_POW_2 * sizeof(unsigned);
	unsigned* root_layer_pow = (unsigned*)malloc(size_root_layer_pow);
	CUDA_CHECK(hipMalloc(&d_root_layer_pow, size_root_layer_pow));

	for (unsigned i = 0; i < 2; i++)
	{
		unsigned st_i = LEN_ROOT_LAYER_POW * i;
		for (unsigned j = 0; j < MAX_LOG; j++)
		{
			unsigned haft_len = 1 << j;
			unsigned st_j = haft_len - 1;
			unsigned ang = 1 << (MAX_LOG - j - 1);
			unsigned wn = i ? root_inv_pow[ang] : root_pow[ang], w = 1;
			for (unsigned k = 0; k < haft_len; k++)
			{
				root_layer_pow[st_i + st_j + k] = w;
				w = mul_mod(w, wn);
			}
		}
	}

	CUDA_CHECK(hipMemcpy(d_root_layer_pow, root_layer_pow, size_root_layer_pow, hipMemcpyHostToDevice));
	free(root_layer_pow);

	unsigned size_packet_product = LEN_PACKET_PRODUCT * sizeof(unsigned);
	unsigned* packet_product = (unsigned*)malloc(size_packet_product);
	CUDA_CHECK(hipMalloc(&d_packet_product, size_packet_product));

	for (unsigned i = 0; i < NUM_OF_PACKET; i++)
	{
		unsigned st = i << (LOG_SYMBOL + 1);
		for (unsigned j = 0; j < SEG_PER_PACKET; j++)
		{
			unsigned k = (i << LOG_SEG) + j;
			packet_product[st + (j << 1)] = sub_mod(0, root_pow[k]);
			packet_product[st + ((j << 1) | 1)] = 1;
			packet_product[st + ((j + SEG_PER_PACKET) << 1)] = sub_mod(0, root_pow[k + SEG_DIFF]);
			packet_product[st + (((j + SEG_PER_PACKET) << 1) | 1)] = 1;
		}
	}
	CUDA_CHECK(hipMemcpy(d_packet_product, packet_product, size_packet_product, hipMemcpyHostToDevice));
	free(packet_product);
	unsigned* tmp;
	CUDA_CHECK(hipMalloc(&tmp, (LEN_ONE_PACKET_PRODUCT << 1) * sizeof(unsigned)));
	for (unsigned i = 0; i < NUM_OF_PACKET; i++)
	{
		unsigned st = i << (LOG_SYMBOL + 1);
		h_build_product(d_packet_product + st, tmp, tmp + LEN_ONE_PACKET_PRODUCT, 1, LOG_SYMBOL + 1, NULL);
	}
	CUDA_CHECK(hipFree(tmp));
	free(inv);
	free(root_pow);
	free(root_inv_pow);

	CUDA_CHECK(hipDeviceSynchronize());
	std::cout << "Init process completed!" << std::endl;

}

void fin()
{
	// clear cuda memory

	CUDA_CHECK(hipDeviceSynchronize());

	for (unsigned i = 0; i < MAX_ACTIVE_ENCODE; i++) {
		CUDA_CHECK(hipHostFree(h_encode_p_slot[i]));
		CUDA_CHECK(hipHostFree(h_encode_y_slot[i]));
	}

	for (unsigned i = 0; i < MAX_ACTIVE_DECODE; i++) {
		CUDA_CHECK(hipHostFree(h_decode_x_slot[i]));
		CUDA_CHECK(hipHostFree(h_decode_y_slot[i]));
		CUDA_CHECK(hipHostFree(h_decode_p_slot[i]));
	}

	free(h_encode_p_slot);
	free(h_encode_y_slot);
	free(h_decode_x_slot);
	free(h_decode_y_slot);
	free(h_decode_p_slot);

	CUDA_CHECK_LAST();

	CUDA_CHECK(hipDeviceReset());
	CUDA_CHECK(hipDeviceSynchronize());

	CUDA_CHECK_LAST();

}

void test_fnt();

void test_poly_mul();

void test_build_init_product();

void test_encode_decode();

void test_fnt_performance();

void test_encode_decode_performance();

int main()
{

	init();

	//test_fnt();
	//
	//test_poly_mul();
	//
	//test_build_init_product();
	//
	//test_encode_decode();
	//
	//test_fnt_performance();

	test_encode_decode_performance();

	fin();

	return 0;
}

void test_fnt() {

	// test correctness of fnt()

	unsigned N_test = 32;

	for (unsigned tt = 0; tt < N_test; tt++) {
		unsigned log_nc = 15, log_nv = 16, nc = 1 << log_nc, nv = 1 << log_nv;
		unsigned size_nc = nc * sizeof(unsigned), size_nv = nv * sizeof(unsigned);
		std::vector<unsigned> c1(nc), c2(nc);
		unsigned* d_c1, * d_c2, * d_v;
		hipMalloc(&d_c1, size_nc);
		hipMemset(d_c1, 0, size_nc);
		hipMalloc(&d_c2, size_nv);
		hipMemset(d_c2, 0, size_nv);
		hipMalloc(&d_v, size_nv);
		hipMemset(d_v, 0, size_nv);

		for (unsigned i = 0; i < nc; i++)
			c1[i] = rand() % (MOD - 1);
		shuffle(c1.begin(), c1.end(), std::default_random_engine(time(NULL)));
		hipMemcpy(d_c1, c1.data(), size_nc, hipMemcpyHostToDevice);

		fnt(d_c1, d_v, log_nc, log_nv, 0, d_N_pos, d_root_layer_pow, d_inv, NULL);
		fnt(d_v, d_c2, log_nv, log_nv, 3, d_N_pos, d_root_layer_pow, d_inv, NULL);

		hipMemcpy(c2.data(), d_c2, size_nc, hipMemcpyDeviceToHost);
		for (unsigned i = 0; i < nc; i++)
			assert(c1[i] == c2[i]);

		hipFree(d_c1);
		hipFree(d_c2);
		hipFree(d_v);

		//std::cout << "FNT test " << tt << " passed!" << std::endl;

	}

	std::cout << "FNT test passed!" << std::endl;

	CUDA_CHECK_LAST();

}

void test_build_init_product() {

	// first 10 element..
	std::vector<unsigned> a1 = { 64375, 0, 52012, 0, 2347, 0, 23649, 0, 30899, 0 }, b1(10);
	hipMemcpy(b1.data(), d_packet_product, 10 * sizeof(unsigned), hipMemcpyDeviceToHost);

	for (unsigned i = 0; i < 10; i++)
		assert(a1[i] == b1[i]);

	// first 10 element of next packet..
	std::vector<unsigned> a2 = { 64375, 0, 31561, 0, 12153, 0, 31103, 0, 20714, 0 }, b2(10);
	hipMemcpy(b2.data(), d_packet_product + (1 << (LOG_SYMBOL + 1)), 10 * sizeof(unsigned), hipMemcpyDeviceToHost);

	for (unsigned i = 0; i < 10; i++)
		assert(a2[i] == b2[i]);

	std::cout << "Test packet_product passed!" << std::endl;

	CUDA_CHECK_LAST();

}

void test_poly_mul() {

	// test correctness of poly_mul()

	srand(time(NULL));

	unsigned N_test = 32;

	for (unsigned tt = 0; tt < N_test; tt++) {

		unsigned log_n = 11;
		unsigned n = 1 << log_n, size_n = n * sizeof(unsigned);

		std::vector<unsigned> a(n), b(n), c1(n << 1, 0), c2(n << 1, 0);

		for (unsigned i = 0; i < n; i++) {
			a[i] = rand() % (MOD - 1); // 2 bytes
			b[i] = rand() % (MOD - 1); // 2 bytes
		}

		unsigned* t1, * t2, * d_c;
		hipMalloc(&t1, size_n << 1);
		hipMalloc(&t2, size_n << 1);
		hipMalloc(&d_c, size_n << 1);
		hipMemcpy(d_c, a.data(), size_n, hipMemcpyHostToDevice);
		hipMemcpy(d_c + n, b.data(), size_n, hipMemcpyHostToDevice);
		h_poly_mul(d_c, d_c + n, t1, t2, d_c, log_n, d_N_pos, d_root_layer_pow, d_inv, NULL);

		for (unsigned i = 0; i < n; i++)
			for (unsigned j = 0; j < n; j++)
				c1[i + j] = add_mod(c1[i + j], mul_mod(a[i], b[j]));

		/*unsigned* d_a, * d_b;
		cudaMalloc(&d_a, size_n);
		cudaMalloc(&d_b, size_n);
		cudaMemcpy(d_a, a.data(), size_n, cudaMemcpyHostToDevice);
		cudaMemcpy(d_b, b.data(), size_n, cudaMemcpyHostToDevice);
		poly_mul_wrapper CUDA_KERNEL(1, 1, NULL, NULL)(d_a, d_b, t1, t2, d_c, log_n, d_N_pos, d_root_layer_pow, d_inv);*/

		hipMemcpy(c2.data(), d_c, size_n << 1, hipMemcpyDeviceToHost);

		for (unsigned i = 0; i < (n << 1); i++)
			assert(c1[i] == c2[i]);

		//std::cout << "Poly mul test " << tt << " passed!" << std::endl;

		hipFree(t1);
		hipFree(t2);
		hipFree(d_c);

	}

	std::cout << "Poly mul test passed!" << std::endl;

	CUDA_CHECK_LAST();

}

void test_encode_decode() {

	// test correctness of encode() and decode()

	srand(time(NULL));

	unsigned N_test = 32;

	for (unsigned tt = 0; tt < N_test; tt++) {
		std::vector<unsigned> a(NUM_OF_NEED_SYMBOL), b(NUM_OF_NEED_SYMBOL << 1), c(NUM_OF_NEED_SYMBOL);

		for (unsigned i = 0; i < NUM_OF_NEED_SYMBOL; i++)
			a[i] = rand() % (MOD - 1); // 2 bytes

		h_encode(a.data(), b.data());
		hipDeviceSynchronize();

		std::vector<unsigned> x(NUM_OF_NEED_SYMBOL), y(NUM_OF_NEED_SYMBOL);

		for (unsigned i = 0; i < NUM_OF_NEED_PACKET; i++) {
			unsigned stx = i * SYMBOL_PER_PACKET;
			for (unsigned j = 0; j < SEG_PER_PACKET; j++) {
				x[stx + j] = stx + j;
				x[stx + j + SEG_PER_PACKET] = stx + j + SEG_DIFF;
				y[stx + j] = b[stx + j];
				y[stx + j + SEG_PER_PACKET] = b[stx + j + SEG_DIFF];
			}
		}

		h_decode(x.data(), y.data(), c.data());
		hipDeviceSynchronize();

		for (unsigned i = 0; i < NUM_OF_NEED_SYMBOL; i++)
			assert(a[i] == c[i]);
		//std::cout << "Encode decode test " << tt << " passed!" << std::endl;
	}

	std::cout << "Encode decode test passed!" << std::endl;

	CUDA_CHECK_LAST();

}

void test_fnt_performance() {

	// fnt() performance with memory already prepare in device

	using namespace std;

	const unsigned N_test = 1024 * 1024 / 64;
	//const unsigned N_test = 1; // use when need profile one..
	unsigned log_n = 16, n = 1 << log_n;
	unsigned size_n = n * sizeof(unsigned);
	vector<vector<unsigned>> a(N_test, vector<unsigned>(n));
	hipStream_t stream[N_test];
	vector<unsigned*> d_a(N_test), d_b(N_test);

	for (unsigned tt = 0; tt < N_test; tt++) {
		for (unsigned i = 0; i < n; i++)
			a[tt][i] = rand() % (MOD - 1);
		CUDA_CHECK(hipStreamCreate(&stream[tt]));
		CUDA_CHECK(hipMallocAsync(&d_a[tt], size_n, stream[tt]));
		CUDA_CHECK(hipMallocAsync(&d_b[tt], size_n, stream[tt]));
		CUDA_CHECK(hipMemcpyAsync(d_a[tt], a[tt].data(), size_n, hipMemcpyHostToDevice, stream[tt]));
	}

	CUDA_CHECK(hipDeviceSynchronize());

	cout << "FNT test start" << endl;

	auto start = chrono::high_resolution_clock::now();

	for (unsigned tt = 0; tt < N_test; tt++) {
		//cudaStreamCreate(&stream[tt]);
		//cudaMallocAsync(&d_a[tt], size_n, stream[tt]);
		//cudaMallocAsync(&d_b[tt], size_n, stream[tt]);
		//cudaMemcpyAsync(d_a[tt], a[tt].data(), size_n, cudaMemcpyHostToDevice, stream[tt]);
		fnt(d_a[tt], d_b[tt], log_n, log_n, 0, d_N_pos, d_root_layer_pow, d_inv, stream[tt]);
		CUDA_CHECK_LAST();
		//cudaFreeAsync(d_a[tt], stream[tt]);
		//cudaFreeAsync(d_b[tt], stream[tt]);
		//cudaStreamDestroy(stream[tt]);
	}

	CUDA_CHECK(hipDeviceSynchronize());
	auto stop = chrono::high_resolution_clock::now();
	auto duration = chrono::duration_cast<chrono::milliseconds>(stop - start).count();

	cout << "FNT " << N_test << " chunks in " << duration << "ms" << endl;

	for (unsigned tt = 0; tt < N_test; tt++) {
		CUDA_CHECK(hipFreeAsync(d_a[tt], stream[tt]));
		CUDA_CHECK(hipFreeAsync(d_b[tt], stream[tt]));
		CUDA_CHECK(hipStreamDestroy(stream[tt]));
	}

}

void test_encode_decode_performance() {

	// test encode(), decode() performance full flow (without prepare memory in device)

	using namespace std;
	srand(time(NULL));

	const unsigned N_test = 128 * 1024 / 64;
	//const unsigned N_test = 1; // use when need profile one..
	const long long symbol_bytes = 2;
	const double size_test_gb = 1.0 * symbol_bytes * NUM_OF_NEED_SYMBOL * N_test / (1024 * 1024 * 1024);

	vector<vector<unsigned>> a(N_test, vector<unsigned>(NUM_OF_NEED_SYMBOL));
	vector<vector<unsigned>> b(N_test, vector<unsigned>(NUM_OF_NEED_SYMBOL << 1));
	vector<vector<unsigned>> c(N_test, vector<unsigned>(NUM_OF_NEED_SYMBOL));

	for (unsigned tt = 0; tt < N_test; tt++)
		for (unsigned i = 0; i < NUM_OF_NEED_SYMBOL; i++)
			a[tt][i] = rand() % (MOD - 1); // 2 bytes

	CUDA_CHECK(hipDeviceSynchronize());

	cout << "Encode performance test start" << endl;

	auto start1 = chrono::high_resolution_clock::now();

	for (unsigned tt = 0; tt < N_test; tt++) {
		h_encode(a[tt].data(), b[tt].data());
		CUDA_CHECK_LAST();
	}

	CUDA_CHECK(hipDeviceSynchronize());
	auto stop1 = chrono::high_resolution_clock::now();
	auto duration1 = chrono::duration_cast<chrono::milliseconds>(stop1 - start1).count();

	cout << "Encode " << N_test << " 64kb chunks in " << duration1 << "ms" << endl;
	cout << "Encode " << (1.0 * size_test_gb) / (1.0 * duration1 / 1000.0) << " GB/s" << endl;

	vector<vector<unsigned>> x(N_test, vector<unsigned>(NUM_OF_NEED_SYMBOL));
	vector<vector<unsigned>> y(N_test, vector<unsigned>(NUM_OF_NEED_SYMBOL));

	for (unsigned tt = 0; tt < N_test; tt++) {
		for (unsigned i = 0; i < NUM_OF_NEED_PACKET; i++) {
			unsigned stx = i * SYMBOL_PER_PACKET;
			for (unsigned j = 0; j < SEG_PER_PACKET; j++) {
				x[tt][stx + j] = stx + j;
				x[tt][stx + j + SEG_PER_PACKET] = stx + j + SEG_DIFF;
				y[tt][stx + j] = b[tt][stx + j];
				y[tt][stx + j + SEG_PER_PACKET] = b[tt][stx + j + SEG_DIFF];
			}
		}
	}

	CUDA_CHECK(hipDeviceSynchronize());

	cout << "Decode performance test start" << endl;

	auto start2 = chrono::high_resolution_clock::now();

	for (unsigned tt = 0; tt < N_test; tt++) {
		h_decode(x[tt].data(), y[tt].data(), c[tt].data());
		CUDA_CHECK_LAST();
	}

	CUDA_CHECK(hipDeviceSynchronize());
	auto stop2 = chrono::high_resolution_clock::now();
	auto duration2 = chrono::duration_cast<chrono::milliseconds>(stop2 - start2).count();

	cout << "Decode " << N_test << " 64kb chunks in " << duration2 << "ms" << endl;
	cout << "Decode " << (1.0 * size_test_gb) / (1.0 * duration2 / 1000.0) << " GB/s" << endl;

	for (unsigned tt = 0; tt < N_test; tt++) {
		for (unsigned i = 0; i < c[tt].size(); i++)
			assert(a[tt][i] == c[tt][i]);
	}

}
